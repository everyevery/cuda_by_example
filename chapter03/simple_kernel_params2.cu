
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

int main () {
	int c;
	int *dev_c;

	hipError_t err;
	err = hipMalloc( (void**)&dev_c, sizeof(int));
	if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                __FILE__, __LINE__ );
        exit( EXIT_FAILURE );
    }

    add<<<1,1>>>(2,7, dev_c);

    err = hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("2 + 7 = %d\n", c);
    hipFree(dev_c);

    return 0;
}